#include <load.h>


int load_file(Context* cont)
{
    //int **Points;
    FILE *fichier = NULL;
    char *chaine = NULL;
    size_t longueur = 0;
    int i = 0, j = 0;
    fichier = fopen(cont->file_name,"r");
    if (fichier != NULL)
    { 
        int compteur = 1;
        while ((getline(&chaine, &longueur, fichier)) != -1)  //chargement des points donnés dans le fichier d'entrée
        { //Attention a la taille limite d'un fichier sur le systeme  
            sscanf(chaine,"%d %d",&i,&j);
            if (compteur==1) // taille du repère
               cont->l = i, cont->h = j;
            else if (compteur == 2) // nombre de points
            {  cont->nb_points = i;
               cont->Points = (int**)malloc(cont->nb_points*sizeof(int *));
	       int k;
               for (k=0; k< cont->nb_points; ++k)
               {
                  cont->Points[k] =(int*) malloc(2*sizeof(int));
                  cont->Points[k][0] = i;
                  cont->Points[k][1] = j;
               }
            }
            else if (compteur-3 < cont->nb_points)
            { 
                  cont->Points[compteur - 3][0] = i;
                  cont->Points[compteur - 3][1] = j;
            }
            compteur += 1;
        }
        fclose(fichier);
    }
    else
       return 1; 

    free(chaine);

    return 0;
	
}

__host__ Context* send_gpu(Context *cont){
    /*
    Alloue la mémoire et copie les données du context CPU -> GPU.
    retourne le pointeur sur le contexte du device
    */

    int k;
    Context *cont_gpu; //device copy of cont
    printf("test 3");
    /* Memory device allocation */
    hipMalloc(&cont_gpu, sizeof(Context));
    /* Memory allocation of Points tab */
    hipMalloc(cont_gpu->Points , cont->nb_points*sizeof(int *));
    for (k=0; k< cont_gpu->nb_points; ++k)
    {
        hipMalloc((void**) cont_gpu->Points[k], 2*sizeof(int));
    }

    printf("test 4");
    /* Copie des données sur GPU */
    hipMemcpy(cont_gpu,cont,sizeof(Context),hipMemcpyHostToDevice);
    hipMemcpy(cont_gpu->Points, cont->Points, cont->nb_points*sizeof(int *),hipMemcpyHostToDevice);
    for (k=0; k< cont_gpu->nb_points; ++k)
    {
        hipMemcpy(cont_gpu->Points[k], cont->Points[k], 2*sizeof(int) ,hipMemcpyHostToDevice);
    }

    printf("test 5");

    return cont_gpu;
}


void dealloc(Context* cont){
    int k;
    for (k=0; k<cont->nb_points; ++k)
        free(cont->Points[k]);
    free(cont->Points);
    free(cont);
}
