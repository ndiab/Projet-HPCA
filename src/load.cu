#include "hip/hip_runtime.h"
#include <load.h>


int load_file(Context* cont)
{
    //int **Points;
    FILE *fichier = NULL;
    char *chaine = NULL;
    size_t longueur = 0;
    int i = 0, j = 0;
    fichier = fopen(cont->file_name,"r");
    if (fichier != NULL)
    { 
        int compteur = 1;
        while ((getline(&chaine, &longueur, fichier)) != -1)  //chargement des points donnés dans le fichier d'entrée
        { //Attention a la taille limite d'un fichier sur le systeme  
            sscanf(chaine,"%d %d",&i,&j);
            if (compteur==1) // taille du repère
               cont->l = i, cont->h = j;
            else if (compteur == 2) // nombre de points
            {  cont->nb_points = i;
               cont->Points = (int**)malloc(cont->nb_points*sizeof(int *));
	       int k;
               for (k=0; k< cont->nb_points; ++k)
               {
                  cont->Points[k] =(int*) malloc(2*sizeof(int));
                  cont->Points[k][0] = i;
                  cont->Points[k][1] = j;
               }
            }
            else if (compteur-3 < cont->nb_points)
            { 
                  cont->Points[compteur - 3][0] = i;
                  cont->Points[compteur - 3][1] = j;
            }
            compteur += 1;
        }
        fclose(fichier);
    }
    else
       return 1; 

    free(chaine);

    return 0;
	
}

__host__ Context* d_load(Context *h_cont){
    /*
    Alloue la mémoire et copie les données du context CPU -> GPU.
    retourne le pointeur sur le contexte du device
    */

    Context *d_cont, *l_cont; //device copy of cont
    l_cont = (Context*)malloc(sizeof(Context));
    int **d_points, **l_points, k;
    l_points = (int**)malloc(h_cont->nb_points*sizeof(int *));

    for(k = 0; k < h_cont->nb_points; k++){
        int* point;
        hipMalloc((void**) &point, 2*sizeof(int));
        hipMemcpy(point, h_cont->Points[k], 2*sizeof(int), hipMemcpyHostToDevice);
        l_points[k] = point;
    }

    hipMalloc((void**) &d_points, h_cont->nb_points*sizeof(int *));
    hipMemcpy(d_points, l_points, h_cont->nb_points*sizeof(int *), hipMemcpyHostToDevice);
    memcpy(l_cont, h_cont, sizeof(Context));
    l_cont->Points = d_points;

    hipMalloc((void**) &d_cont, sizeof(Context));
    hipMemcpy(d_cont, l_cont, sizeof(Context), hipMemcpyHostToDevice);
    

    return d_cont;
}

__global__ void d_free(Context* d_cont){
    int k;
    for (k=0; k< d_cont->nb_points; ++k)
       free(d_cont->Points[0]);
    free(d_cont->Points);
    free(d_cont);
}


void h_free(Context* cont){
    int k;
    for (k=0; k<cont->nb_points; ++k)
        free(cont->Points[k]);
    free(cont->Points);
    free(cont);
}
