#include "hip/hip_runtime.h"
#include <algo2.h>


unsigned long long int cpu2(Context *cont){
    int i,j;
    unsigned long long y_min, surface_max = 0; 
    for (i=0;i<cont->nb_points-1;++i)
        for (j=i+1;j<cont->nb_points;++j)
        {
           if (j==i+1)
              y_min = cont->h;
           else
              y_min = MIN(y_min, cont->Points[j-1][1]);
           surface_max = MAX(surface_max,y_min*(cont->Points[j][0] - cont->Points[i][0]));
         }  
    return surface_max;
}

__global__ void kernel_par_2(Context* cont){
    __shared__ int s_max[NB_THREADS];  // tableau contenant le surface_max de chaque thread
    int index = threadIdx.x + blockIdx.x * blockDim.x ;
    int nb_iter = 1; // cont->nb_points / NB_THREADS;
    int pos = nb_iter * index + (MIN(cont->nb_points%NB_THREADS, index));
    if (cont->nb_points % NB_THREADS >= threadIdx.x) //Cas ou le nombre de points n'est pas un multiple de notre NB_THREADS
        nb_iter++;
    //CALCUL
    int i, j, y_min;
    unsigned long long surface_max = 0;
    int cpt = 0;

  //  printf("Lancement du thread no %d\n", threadIdx.x);

    for (i=0;i<nb_iter;++i){
        for (j=pos+1;j<cont->nb_points;++j){
           cpt++;
           if (j==pos+1)
              y_min = cont->h;
           else 
              y_min = MIN(y_min, cont->Points[j-1][1]);
           surface_max = MAX(surface_max,y_min*(cont->Points[j][0] - cont->Points[pos][0]));
        }
        pos++;
    }

  //  printf("Thread no %d finished \n", threadIdx.x);

  //  printf("thread no : %d surface_max = %llu, nb_iter = %d, pos = %d, nb_tour = %d\n",threadIdx.x, surface_max, nb_iter, pos, cpt);
    // stockage des resultats dans la mémoire partagée
    s_max[threadIdx.x] = surface_max;
    __syncthreads();

    /* Comparaison des resultats afin de retrouver la valeur maximale */
    int a = NB_THREADS/2;
    while(a>0 && a>threadIdx.x){
        unsigned long long i = MAX(s_max[2*threadIdx.x], s_max[2*threadIdx.x+1]);
      //  printf("valeur de i = %d\n",i);
        __syncthreads();
        s_max[threadIdx.x] = i;
        __syncthreads();
        a = a >> 1;
    }
    __syncthreads();
    //cont->surface_max = s_max[0];

    if(threadIdx.x == 0)
        atomicMax(&(cont->surface_max), (unsigned long long int)s_max[0]);

}

__host__ unsigned long long int gpu2(Context* cont){

    unsigned long long surface_max = 0;
    Context * d_cont;

    printf("sending to gpu... ");
    d_cont = d_load(cont);
    printf(" sent !\n");

    printf("lancement du kernel : \n");
    cont->start = my_gettimeofday();
    kernel_par_2<<<cont->nb_points/NB_THREADS,NB_THREADS>>>(d_cont);
    printf("sortie du kernel\n");

    hipMemcpy(&surface_max, &(d_cont->surface_max), sizeof(unsigned long long), hipMemcpyDeviceToHost); // récupération du résultat
    cont->end = my_gettimeofday();

    d_free<<<1,1>>>(d_cont);

    return surface_max;

}



int algo2(Context *cont, int env)
{
    int surface_max = 0;

    switch (env){
        case CPU:
                cont->start = my_gettimeofday();
		surface_max = cpu2(cont);
                cont->end = my_gettimeofday();
		break;
        case GPU:
		printf("lancement du GPU\n");
                surface_max = gpu2(cont);
                break;
    }
  
    return surface_max;
}
