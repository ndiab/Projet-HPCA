#include "hip/hip_runtime.h"
#include <algo1.h>

int cpu1(Context *cont){

    int i,j;
    unsigned long long y_min, surface_max = 0;
    for (i=0;i<cont->nb_points-1;++i)
        for (j=i+1;j<cont->nb_points;++j)
        {
           if (j==i+1)
              y_min = cont->h;
           else if ( j-1 == i+1) // 1 seul point separe xi et xj (i.e: xi < pt <xj)
              y_min = cont->Points[i+1][1];
           else // au moins deux points separe xi et xj (i.e: xi< p1<p2 <xj)
           { 
              y_min = cont->Points[i+1][1];
	      int k;
              for (k=i+1;k<=j-1;++k)
                  y_min = MIN(y_min, cont->Points[k][1]);
           }
           surface_max = MAX(surface_max,y_min*(cont->Points[j][0] - cont->Points[i][0]));
           if(y_min*(cont->Points[j][0] - cont->Points[i][0]) == 54158500)
		printf("pos = %d, j = %d ", i, j);
	   if(y_min*(cont->Points[j][0] - cont->Points[i][0]) == 6741)
		printf("pos = %d, j = %d ", i, j);
        }

    return surface_max;

}

__global__ void kernel_seq_1(Context *cont){
    //int index = threadIdx.x + blockIdx.x * blockDim.x;

    int i, j;
    unsigned long long y_min;
    cont->surface_max = 0;
    for (i=0;i<cont->nb_points-1;++i)
        for (j=i+1;j<cont->nb_points;++j)
        {
           if (j==i+1)
              y_min = cont->h;
           else if ( j-1 == i+1) // 1 seul point separe xi et xj (i.e: xi < pt <xj)
              y_min = cont->Points[i+1][1];
           else // au moins deux points separe xi et xj (i.e: xi< p1<p2 <xj)
           { 
              y_min = cont->Points[i+1][1];
	      int k;
              for (k=i+1;k<=j-1;++k)
                  y_min = MIN(y_min, cont->Points[k][1]);
           }
           cont->surface_max = MAX(cont->surface_max,y_min*(cont->Points[j][0] - cont->Points[i][0]));
	   
        }
}


__device__ void d_max(int* s_max){
    int i = MAX(s_max[2*threadIdx.x], s_max[2*threadIdx.x+1]);
    __syncthreads();
    s_max[threadIdx.x] = i;
}


__global__ void kernel_par_1(Context* cont){
    __shared__ int s_max[NB_THREADS];  // tableau contenant le surface_max de chaque thread
    int nb_iter = cont->nb_points / NB_THREADS;
    int pos = nb_iter * threadIdx.x + (MIN(cont->nb_points%NB_THREADS, threadIdx.x));
    if (cont->nb_points % NB_THREADS >= threadIdx.x) //Cas ou le nombre de points n'est pas un multiple de notre NB_THREADS
        nb_iter++;
    //CALCUL
    int i, j, y_min;
    unsigned long long surface_max = 0;
    int cpt = 0;

  //  printf("Lancement du thread no %d\n", threadIdx.x);

    for (i=0;i<nb_iter;++i){
        for (j=pos+1;j<cont->nb_points;++j){
           cpt++;
           if (j==pos+1)
              y_min = cont->h;
           else if ( j-1 == pos+1) // 1 seul point separe xi et xj (i.e: xi < pt <xj)
              y_min = cont->Points[pos+1][1];
           else // au moins deux points separe xi et xj (i.e: xi< p1<p2 <xj)
           { 
              y_min = cont->Points[pos+1][1];
	      int k;
              for (k=pos+1;k<=j-1;++k)
                  y_min = MIN(y_min, cont->Points[k][1]);
           }
           surface_max = MAX(surface_max,y_min*(cont->Points[j][0] - cont->Points[pos][0]));
        } 
	pos++;
     }

  //  printf("Thread no %d finished \n", threadIdx.x);

  //  printf("thread no : %d surface_max = %llu, nb_iter = %d, pos = %d, nb_tour = %d\n",threadIdx.x, surface_max, nb_iter, pos, cpt);
    // stockage des resultats dans la mémoire partagée
    s_max[threadIdx.x] = surface_max;
    __syncthreads();

    /* Comparaison des resultats afin de retrouver la valeur maximale */
    int a = NB_THREADS/2;
    while(a>0 && a>threadIdx.x){
        unsigned long long i = MAX(s_max[2*threadIdx.x], s_max[2*threadIdx.x+1]);
      //  printf("valeur de i = %d\n",i);
        __syncthreads();
        s_max[threadIdx.x] = i;
        __syncthreads();
        a = a >> 1;
    }
    __syncthreads();
    cont->surface_max = s_max[0];

}

__host__ int gpu1(Context* cont){

    unsigned long long surface_max = 0;
    Context * d_cont;
 
    printf("sending to gpu... ");
    d_cont = d_load(cont);
    printf(" sent !\n");

    printf("lancement du kernel : \n");
    cont->start = my_gettimeofday();
    kernel_par_1<<<1,NB_THREADS>>>(d_cont);  
    printf("sortie du kernel\n");

    hipMemcpy(&surface_max, &(d_cont->surface_max), sizeof(unsigned long long), hipMemcpyDeviceToHost); // récupération du résultat
    cont->end = my_gettimeofday(); 

    d_free<<<1,1>>>(d_cont);
    
    return surface_max;

}




int algo1(Context *cont, int env)
{  
    int surface_max = 0;

    switch (env){
        case CPU:
		cont->start = my_gettimeofday(); 
		surface_max = cpu1(cont);
		cont->end = my_gettimeofday();
		break;
        case GPU:
                printf("lancement du GPU\n");
		surface_max = gpu1(cont);
		break;
    }
  
    return surface_max;
}
