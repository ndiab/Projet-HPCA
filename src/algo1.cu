#include "hip/hip_runtime.h"
#include <algo1.h>

int cpu1(Context *cont){

    int i,j, y_min, surface_max = 0;
    for (i=0;i<cont->nb_points-1;++i)
        for (j=i+1;j<cont->nb_points;++j)
        {
           if (j==i+1)
              y_min = cont->h;
           else if ( j-1 == i+1) // 1 seul point separe xi et xj (i.e: xi < pt <xj)
              y_min = cont->Points[i+1][1];
           else // au moins deux points separe xi et xj (i.e: xi< p1<p2 <xj)
           { 
              y_min = cont->Points[i+1][1];
	      int k;
              for (k=i+1;k<=j-1;++k)
                  y_min = MIN(y_min, cont->Points[k][1]);
           }
           surface_max = MAX(surface_max,y_min*(cont->Points[j][0] - cont->Points[i][0]));
        }

    return surface_max;

}

__global__ void kernel_seq_1(Context *cont){
    //int index = threadIdx.x + blockIdx.x * blockDim.x;

    int i, j, y_min;
    cont->surface_max = 0;
    for (i=0;i<cont->nb_points-1;++i)
        for (j=i+1;j<cont->nb_points;++j)
        {
           if (j==i+1)
              y_min = cont->h;
           else if ( j-1 == i+1) // 1 seul point separe xi et xj (i.e: xi < pt <xj)
              y_min = cont->Points[i+1][1];
           else // au moins deux points separe xi et xj (i.e: xi< p1<p2 <xj)
           { 
              y_min = cont->Points[i+1][1];
	      int k;
              for (k=i+1;k<=j-1;++k)
                  y_min = MIN(y_min, cont->Points[k][1]);
           }
           cont->surface_max = MAX(cont->surface_max,y_min*(cont->Points[j][0] - cont->Points[i][0]));
        }
}


__device__ void d_max(int* s_max){
    int i = MAX(s_max[2*threadIdx.x], s_max[2*threadIdx.x+1]);
    __syncthreads();
    s_max[threadIdx.x] = i;
}


__global__ void kernel_par_1(Context* cont){
    __shared__ int s_max[NB_THREADS];  // tableau contenant le surface_max de chaque thread
    int nb_iter = cont->nb_points / NB_THREADS;
    int pos = nb_iter * threadIdx.x + (MIN(cont->nb_points%NB_THREADS, threadIdx.x));
    if (cont->nb_points % NB_THREADS >= threadIdx.x) //Cas ou le nombre de points n'est pas un multiple de notre NB_THREADS
        nb_iter++;
    //CALCUL
    int i, j, y_min;
    int surface_max = 0;
    int cpt = 0;

    for (i=0;i<nb_iter;++i){
        for (j=pos+1;j<cont->nb_points;++j){
           cpt++;
           if (j==pos+1)
              y_min = cont->h;
           else if ( j-1 == pos+1) // 1 seul point separe xi et xj (i.e: xi < pt <xj)
              y_min = cont->Points[pos+1][1];
           else // au moins deux points separe xi et xj (i.e: xi< p1<p2 <xj)
           { 
              y_min = cont->Points[pos+1][1];
	      int k;
              for (k=pos+1;k<=j-1;++k)
                  y_min = MIN(y_min, cont->Points[k][1]);
           }
           surface_max = MAX(cont->surface_max,y_min*(cont->Points[j][0] - cont->Points[pos][0]));
        } 
    }

   // printf("thread no : %d surface_max = %d, nb_iter = %d, pos = %d, nb_tour = %d\n",threadIdx.x, surface_max, nb_iter, pos, cpt);
    // stockage des resultats dans la mémoire partagée
    s_max[threadIdx.x] = surface_max;
    __syncthreads();

    /* Comparaison des resultats afin de retrouver la valeur maximale */
    int a = NB_THREADS/2;
    while(a>0 && a>threadIdx.x){
        int i = MAX(s_max[2*threadIdx.x], s_max[2*threadIdx.x+1]);
      //  printf("valeur de i = %d\n",i);
        __syncthreads();
        s_max[threadIdx.x] = i;
        __syncthreads();
        a = a >> 1;
    }
    __syncthreads();
    cont->surface_max = s_max[0];

}

__host__ int gpu1(Context* cont){

    int surface_max = 0;
    Context * d_cont;

    d_cont = d_load(cont);

    kernel_par_1<<<1,NB_THREADS>>>(d_cont); //premiers tests en sequentiel sur GPU   
     
    hipMemcpy(&surface_max, &(d_cont->surface_max), sizeof(int), hipMemcpyDeviceToHost); // récupération du résultat

    d_free<<<1,1>>>(d_cont);
    
    return surface_max;

}




int algo1(Context *cont, int env)
{  
    int surface_max = 0;

    switch (env){
        case CPU:
		surface_max = cpu1(cont);
		break;
        case GPU:
		surface_max = gpu1(cont);
		break;
    }
  
    return surface_max;
}
