#include "hip/hip_runtime.h"
#include <algo1.h>

unsigned long long int h_algo1(Context *cont){

    int i,j;
    unsigned long long int y_min, surface_max = 0;
    for (i=0;i<cont->nb_points-1;++i)
        for (j=i+1;j<cont->nb_points;++j)
        {
           if (j==i+1)
              y_min = cont->h;
           else if ( j-1 == i+1) // 1 seul point separe xi et xj (i.e: xi < pt <xj)
              y_min = cont->Points[i+1][1];
           else // au moins deux points separe xi et xj (i.e: xi< p1<p2 <xj)
           { 
              y_min = cont->Points[i+1][1];
	      int k;
              for (k=i+1;k<=j-1;++k)
                  y_min = MIN(y_min, cont->Points[k][1]);
           }
           surface_max = MAX(surface_max,y_min*(cont->Points[j][0] - cont->Points[i][0]));
           if(y_min*(cont->Points[j][0] - cont->Points[i][0]) == 54158500)
		printf("pos = %d, j = %d ", i, j);
	   if(y_min*(cont->Points[j][0] - cont->Points[i][0]) == 6741)
		printf("pos = %d, j = %d ", i, j);
        }

    return surface_max;

}

__device__ void d_max(int* s_max){
    int i = MAX(s_max[2*threadIdx.x], s_max[2*threadIdx.x+1]);
    __syncthreads();
    s_max[threadIdx.x] = i;
}


__global__ void kernel_1(Context* cont){
    __shared__ unsigned long long int s_max[NB_THREADS];  // tableau contenant le surface_max de chaque thread
    int index = threadIdx.x + blockIdx.x * blockDim.x ;
    int nb_iter = 1; //cont->nb_points / NB_THREADS;
    int pos = nb_iter * index + (MIN(cont->nb_points%NB_THREADS, index));
    if (cont->nb_points % NB_THREADS >= threadIdx.x) //Cas ou le nombre de points n'est pas un multiple de notre NB_THREADS
        nb_iter++;
    //CALCUL
    int i, j, y_min;
    unsigned long long int surface_max = 0;
    int cpt = 0;

  //  printf("Lancement du thread no %d\n", threadIdx.x);

    for (i=0;i<nb_iter;++i){
        for (j=pos+1;j<cont->nb_points;++j){
           cpt++;
           if (j==pos+1)
              y_min = cont->h;
           else if ( j-1 == pos+1) // 1 seul point separe xi et xj (i.e: xi < pt <xj)
              y_min = cont->Points[pos+1][1];
           else // au moins deux points separe xi et xj (i.e: xi< p1<p2 <xj)
           { 
              y_min = cont->Points[pos+1][1];
	      int k;
              for (k=pos+1;k<=j-1;++k)
                  y_min = MIN(y_min, cont->Points[k][1]);
           }
           surface_max = MAX(surface_max,y_min*(cont->Points[j][0] - cont->Points[pos][0]));
        } 
	pos++;
    }

  //  printf("Thread no %d finished \n", threadIdx.x);

  //  printf("thread no : %d surface_max = %llu, nb_iter = %d, pos = %d, nb_tour = %d\n",threadIdx.x, surface_max, nb_iter, pos, cpt);
    // stockage des resultats dans la mémoire partagée
    s_max[threadIdx.x] = surface_max;
    __syncthreads();

    /* Comparaison des resultats afin de retrouver la valeur maximale */
    int a = NB_THREADS/2;
    while(a>0 && a>threadIdx.x){
        unsigned long long int i = MAX(s_max[2*threadIdx.x], s_max[2*threadIdx.x+1]);
      //  printf("valeur de i = %d\n",i);
        __syncthreads();
        s_max[threadIdx.x] = i;
        __syncthreads();
        a = a >> 1;
    }
    __syncthreads();

    if(threadIdx.x == 0)
        atomicMax(&(cont->surface_max), (unsigned long long int)s_max[0]);

}

__host__ unsigned long long int d_algo1(Context* cont){

    unsigned long long int surface_max = 0;
    Context * d_cont;
 
    printf("sending to gpu... ");
    d_cont = d_load(cont);
    printf(" sent !\n");

    printf("lancement du kernel : \n");
    cont->start = my_gettimeofday();
    kernel_1<<<cont->nb_points/NB_THREADS,NB_THREADS>>>(d_cont);  
    printf("sortie du kernel\n");

    hipMemcpy(&surface_max, &(d_cont->surface_max), sizeof(unsigned long long int), hipMemcpyDeviceToHost); // récupération du résultat
    cont->end = my_gettimeofday(); 

    d_free<<<1,1>>>(d_cont);
    
    return surface_max;

}




unsigned long long int algo1(Context *cont, int env)
{  
    int surface_max = 0;

    switch (env){
        case CPU:
		cont->start = my_gettimeofday(); 
		surface_max = h_algo1(cont);
		cont->end = my_gettimeofday();
		break;
        case GPU:
                printf("lancement du GPU\n");
		surface_max = d_algo1(cont);
		break;
    }
  
    return surface_max;
}
