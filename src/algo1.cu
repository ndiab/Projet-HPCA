#include "hip/hip_runtime.h"
#include <algo1.h>

int cpu1(Context *cont){

    int i,j, y_min, surface_max = 0;
    for (i=0;i<cont->nb_points-1;++i)
        for (j=i+1;j<cont->nb_points;++j)
        {
           if (j==i+1)
              y_min = cont->h;
           else if ( j-1 == i+1) // 1 seul point separe xi et xj (i.e: xi < pt <xj)
              y_min = cont->Points[i+1][1];
           else // au moins deux points separe xi et xj (i.e: xi< p1<p2 <xj)
           { 
              y_min = cont->Points[i+1][1];
	      int k;
              for (k=i+1;k<=j-1;++k)
                  y_min = MIN(y_min, cont->Points[k][1]);
           }
           surface_max = MAX(surface_max,y_min*(cont->Points[j][0] - cont->Points[i][0]));
        }

    return surface_max;

}


__host__ int gpu1(Context* cont){

    int surface_max = 0;
    //Context *cont; //device copy of cont
    
     
    
    return surface_max;

}

__global__ void kernel1(Context *cont){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
}


int algo1(Context *cont, int env)
{  
    int surface_max = 0;

    switch (env){
        case CPU:
		surface_max = cpu1(cont);
		break;
        case GPU:
		surface_max = gpu1(cont);
		break;
    }
  
    return surface_max;
}
