#include "hip/hip_runtime.h"
#include <algo1.h>

int cpu1(Context *cont){

    int i,j, y_min, surface_max = 0;
    for (i=0;i<cont->nb_points-1;++i)
        for (j=i+1;j<cont->nb_points;++j)
        {
           if (j==i+1)
              y_min = cont->h;
           else if ( j-1 == i+1) // 1 seul point separe xi et xj (i.e: xi < pt <xj)
              y_min = cont->Points[i+1][1];
           else // au moins deux points separe xi et xj (i.e: xi< p1<p2 <xj)
           { 
              y_min = cont->Points[i+1][1];
	      int k;
              for (k=i+1;k<=j-1;++k)
                  y_min = MIN(y_min, cont->Points[k][1]);
           }
           surface_max = MAX(surface_max,y_min*(cont->Points[j][0] - cont->Points[i][0]));
        }

    return surface_max;

}

__global__ void kernel1(Context *cont){
    //int index = threadIdx.x + blockIdx.x * blockDim.x;

    int i, j, y_min;
    cont->surface_max = 0;
    for (i=0;i<cont->nb_points-1;++i)
        for (j=i+1;j<cont->nb_points;++j)
        {
           if (j==i+1)
              y_min = cont->h;
           else if ( j-1 == i+1) // 1 seul point separe xi et xj (i.e: xi < pt <xj)
              y_min = cont->Points[i+1][1];
           else // au moins deux points separe xi et xj (i.e: xi< p1<p2 <xj)
           { 
              y_min = cont->Points[i+1][1];
	      int k;
              for (k=i+1;k<=j-1;++k)
                  y_min = MIN(y_min, cont->Points[k][1]);
           }
           cont->surface_max = MAX(cont->surface_max,y_min*(cont->Points[j][0] - cont->Points[i][0]));
        }
}

__host__ int gpu1(Context* cont){

    int surface_max = 0;
    Context * d_cont;

    d_cont = send_gpu(cont);

    kernel1<<<1,1>>>(d_cont); //premiers tests en sequentiel sur GPU   
     
    hipMemcpy(&surface_max, &(d_cont->surface_max), sizeof(int), hipMemcpyDeviceToHost);
    
    return surface_max;

}




int algo1(Context *cont, int env)
{  
    int surface_max = 0;

    switch (env){
        case CPU:
		surface_max = cpu1(cont);
		break;
        case GPU:
		surface_max = gpu1(cont);
		break;
    }
  
    return surface_max;
}
