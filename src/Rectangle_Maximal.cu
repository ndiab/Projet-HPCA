#include "hip/hip_runtime.h"
#include <Rectangle_Maximal.h>

/*
__global__ void kernel1(unsigned int *Points,unsigned int size)
{
unsigned int i = threadIdx.x
unsigned int j = threaddy.y
}
*/



int main(int argc, char* argv[])
{

    /* 1 - Initialisation du contexte */  
    Context *cont = (Context*) malloc(sizeof(Context));
    cont->file_name = argv[1];
    
    if (load_file(cont)){
	printf("Problème de chargement du fichier");
	return 1;
    }
	

    /* 2 - Lancement de l'algorithme demandé pour résoudre le problème */ 
    int Surface_max = 0;
    int env = atoi(argv[3]);

    switch (atoi(argv[2]))
    {
	case 1 :
	    Surface_max = algo1(cont, env);
	    break;
	case 2 :
	    Surface_max = algo2(cont, env);
	    break;
    }


    /* 3 - Affichage du resultat */
    printf("Surface maximale = %d\n", Surface_max);
    
    // deploiement sur GPU
    //dim3 threabParBlock(,,1)
    //dim3 tailleGrille (,,1)

    /* 4 - Libération de la mémoire */
    h_free(cont);

    return 0;
}
