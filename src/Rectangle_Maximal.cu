#include "hip/hip_runtime.h"
#include <Rectangle_Maximal.h>

/*
__global__ void kernel1(unsigned int *Points,unsigned int size)
{
unsigned int i = threadIdx.x
unsigned int j = threaddy.y
}
*/

double my_gettimeofday(){
  struct timeval tmp_time;
  gettimeofday(&tmp_time, NULL);
  return tmp_time.tv_sec + (tmp_time.tv_usec * 1.0e-6L);
}


int main(int argc, char* argv[])
{

    /* 1 - Initialisation du contexte */  
    Context *cont = (Context*) malloc(sizeof(Context));
    cont->file_name = argv[1];
    
    printf("loading...");
    if (load_file(cont)){
	printf("Problème de chargement du fichier");
	return 1;
    }
    printf("   loaded !\n");

    cont-> surface_max = 0;
	
    /* 2 - Lancement de l'algorithme demandé pour résoudre le problème */ 
    unsigned long long int Surface_max = 0;
    int env = atoi(argv[3]);

    switch (atoi(argv[2]))
    {
	case 1 :
            printf("lancement de l'algo\n");
	    Surface_max = algo1(cont, env);
	    break;
	case 2 :
	    Surface_max = algo2(cont, env);
	    break;
    }


    /* 3 - Affichage du resultat */
    printf("Surface maximale = %llu\n", Surface_max);
    
    // deploiement sur GPU
    //dim3 threabParBlock(,,1)
    //dim3 tailleGrille (,,1)

    /* 4 - Libération de la mémoire */
    printf("Temps total de calcul : %g sec\n", cont->end - cont->start);
    h_free(cont);

    return 0;
}
