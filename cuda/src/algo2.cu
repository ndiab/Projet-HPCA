#include "hip/hip_runtime.h"
#include <algo2.h>

/*__device__ void compute(Context* cont, int i, int j){
	
}*/


__global__ void kernel_par_2(Context* cont){
    __shared__ unsigned long long int s_max[NB_THREADS];  // tableau contenant le surface_max de chaque thread
    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int  j;
    //CALCUL
    unsigned long long int surface_max = 0;
    unsigned long long int y_min;

     // mettre la fonciton compute ici

     for (j=i+1;j<cont->nb_points;++j){
        if (j==i+1)
           y_min = cont->h;
        else 
           y_min = MIN(y_min, cont->Points[j-1][1]);
        surface_max = MAX(surface_max,y_min*(cont->Points[j][0] - cont->Points[i][0]));
     }

    s_max[threadIdx.x] = surface_max;
    __syncthreads();

    unsigned long long int s;
 
    /* Comparaison des resultats afin de retrouver la valeur maximale */
    int a = NB_THREADS >>1 ;
    while(a>0 && a>threadIdx.x){
        s = MAX(s_max[2*threadIdx.x], s_max[2*threadIdx.x+1]);
        s_max[threadIdx.x] = s;
	__syncthreads();
        a = a >> 1;
    }

    if(threadIdx.x == 0)
        atomicMax(&(cont->surface_max), (unsigned long long int)s_max[0]);
	
}

__host__ unsigned long long int gpu2(Context* cont){

    unsigned long long int surface_max = 0;
    Context * d_cont;

    printf("sending to gpu... ");
    d_cont = d_load(cont);
    printf(" sent !\n");

    printf("lancement du kernel : \n");
    cont->start = my_gettimeofday();
    kernel_par_2<<<cont->nb_points/NB_THREADS + 1,NB_THREADS>>>(d_cont);
    printf("sortie du kernel\n");

    hipMemcpy(&surface_max, &(d_cont->surface_max), sizeof(unsigned long long int), hipMemcpyDeviceToHost); // récupération du résultat
    cont->end = my_gettimeofday();

    d_free<<<1,1>>>(d_cont);

    return surface_max;

}



unsigned long long int algo2(Context *cont)
{
    unsigned long long int surface_max = 0;

    printf("lancement du GPU\n");
    surface_max = gpu2(cont);
    
    return surface_max;
}
