#include "hip/hip_runtime.h"
#include <algo2.h>

__global__ void kernel_par_2(Context* cont){
    __shared__ int s_max[NB_THREADS];  // tableau contenant le surface_max de chaque thread
    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int y_min, j;
    //CALCUL
    unsigned long long surface_max = 0;


     for (j=i+1;j<cont->nb_points;++j){
        if (j==i+1)
           y_min = cont->h;
        else 
           y_min = MIN(y_min, cont->Points[j-1][1]);
        surface_max = MAX(surface_max,y_min*(cont->Points[j][0] - cont->Points[i][0]));
     }

    s_max[threadIdx.x] = surface_max;
    __syncthreads();

    /* Comparaison des resultats afin de retrouver la valeur maximale */
    int a = NB_THREADS >>1 ;
    while(a>0 && a>threadIdx.x){
        unsigned long long i = MAX(s_max[2*threadIdx.x], s_max[2*threadIdx.x+1]);
        __syncthreads();
        s_max[threadIdx.x] = i;
        __syncthreads();
        a = a >> 1;
    }
    __syncthreads();

    if(threadIdx.x == 0)
        atomicMax(&(cont->surface_max), (unsigned long long int)s_max[0]);

}

__host__ unsigned long long int gpu2(Context* cont){

    unsigned long long surface_max = 0;
    Context * d_cont;

    printf("sending to gpu... ");
    d_cont = d_load(cont);
    printf(" sent !\n");

    printf("lancement du kernel : \n");
    cont->start = my_gettimeofday();
    kernel_par_2<<<cont->nb_points/NB_THREADS + 1,NB_THREADS>>>(d_cont);
    printf("sortie du kernel\n");

    hipMemcpy(&surface_max, &(d_cont->surface_max), sizeof(unsigned long long), hipMemcpyDeviceToHost); // récupération du résultat
    cont->end = my_gettimeofday();

    d_free<<<1,1>>>(d_cont);

    return surface_max;

}



int algo2(Context *cont, int env)
{
    int surface_max = 0;

    printf("lancement du GPU\n");
    surface_max = gpu2(cont);
    
    return surface_max;
}
