#include "hip/hip_runtime.h"
#include <algo1.h>


__device__ void d_max(int* s_max){
    int i = MAX(s_max[2*threadIdx.x], s_max[2*threadIdx.x+1]);
    __syncthreads();
    s_max[threadIdx.x] = i;
}


__global__ void kernel_1_dim1(Context* cont){
    __shared__ unsigned long long int s_max[NB_THREADS];  // tableau contenant le surface_max de chaque thread
    int i = threadIdx.x + blockIdx.x * blockDim.x ;


    //CALCUL
    int j, y_min;
    unsigned long long int surface_max = 0;

    if(i >= cont-> nb_points)
	s_max[threadIdx.x] = 0;
    else{
	    for (j=i+1;j<cont->nb_points;++j){
	       if (j==i+1)
		  y_min = cont->h;
	       else if ( j-1 == i+1) // 1 seul point separe xi et xj (i.e: xi < pt <xj)
		  y_min = cont->Points[i+1][1];
	       else // au moins deux points separe xi et xj (i.e: xi< p1<p2 <xj)
	       { 
		  y_min = cont->Points[i+1][1];
		  int k;
		  for (k=i+1;k<=j-1;++k)
		      y_min = MIN(y_min, cont->Points[k][1]);
	       }
	       surface_max = MAX(surface_max,y_min*(cont->Points[j][0] - cont->Points[i][0]));
	    }
  
	    // stockage des resultats dans la mémoire partagée
	    s_max[threadIdx.x] = surface_max;
    }
    __syncthreads();

    /* Comparaison des resultats afin de retrouver la valeur maximale */
    int a = NB_THREADS>>1;
    while(a>0 && a>threadIdx.x){
        unsigned long long int i = MAX(s_max[2*threadIdx.x], s_max[2*threadIdx.x+1]);
        __syncthreads();
        s_max[threadIdx.x] = i;
        __syncthreads();
        a = a >> 1;
    }
    __syncthreads();

    if(threadIdx.x == 0)
        atomicMax(&(cont->surface_max), (unsigned long long int)s_max[0]);

}


__global__ void kernel_1_dim2(Context* cont){
    //int nb_threads = blockDim.x * blockDim.y;
    __shared__ unsigned long long int s_max[NB_THREADS];  // tableau contenant le surface_max de chaque thread du bloc courant
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    //CALCUL
    int y_min;
    int pos_shared_tab = threadIdx.x * blockDim.x + threadIdx.y;

    if (j <= i || j>= cont-> nb_points || i >= cont->nb_points){
        s_max[threadIdx.x] =  0;
    }else{

	    if (j==i+1)
	       y_min = cont->h;
	    else if ( j-1 == i+1) // 1 seul point separe xi et xj (i.e: xi < pt <xj)
	       y_min = cont->Points[i+1][1];
	    else if (j > i) // au moins deux points separe xi et xj (i.e: xi< p1<p2 <xj)
	    { 
	       y_min = cont->Points[i+1][1];
	       int k;
	       for (k=i+1;k<=j-1;++k)
		  y_min = MIN(y_min, cont->Points[k][1]);
	     }
	    s_max[pos_shared_tab] =  y_min*(cont->Points[j][0] - cont->Points[i][0]);
    }
    __syncthreads();

    /* Comparaison des resultats afin de retrouver la valeur maximale */
    int a = NB_THREADS >> 1;
    while(a>0 && a> pos_shared_tab){
	unsigned long long int i = MAX(s_max[2*pos_shared_tab], s_max[2*pos_shared_tab+1]);
	__syncthreads();
	s_max[pos_shared_tab] = i;
	__syncthreads();
	a = a >> 1;
    }

    __syncthreads();

   if(threadIdx.x == 0 && threadIdx.y == 0  && s_max[0] != 0)
        atomicMax(&(cont->surface_max), (unsigned long long int)s_max[0]);

}

__host__ unsigned long long int d_algo1(Context* cont){

    unsigned long long int surface_max = 0;
    Context * d_cont;
 
    printf("sending to gpu... ");
    d_cont = d_load(cont);
    printf(" sent !\n");


    dim3 threadsPerBlock(8,8);
    dim3 nbBlocks(cont->nb_points/threadsPerBlock.x + 1, cont->nb_points/threadsPerBlock.y + 1);
    
    printf("lancement du kernel ... \n");
    cont->start = my_gettimeofday();
    kernel_1_dim2<<<nbBlocks, threadsPerBlock>>>(d_cont);


    hipMemcpy(&surface_max, &(d_cont->surface_max), sizeof(unsigned long long int), hipMemcpyDeviceToHost); // récupération du résultat
    cont->end = my_gettimeofday(); 

    d_free<<<1,1>>>(d_cont);
    
    return surface_max;

}




unsigned long long int algo1(Context *cont, int env)
{  
    int surface_max = 0;

    printf("lancement du GPU\n");
    surface_max = d_algo1(cont);

    return surface_max;
}
